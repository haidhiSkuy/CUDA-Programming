#include "hip/hip_runtime.h"

#include <stdio.h> 

__global__ void helloFromGPU()
{ 
    printf("Hello CUDA\n"); 
}

int main() {
    
    dim3 grid(8); 
    dim3 block(4);

    helloFromGPU<<<grid, block>>>();
    
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}