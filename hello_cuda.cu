#include "hip/hip_runtime.h"

#include <stdio.h> 

__global__ void helloFromGPU()
{ 
    printf("Hello CUDA\n"); 
}

int main() {
    
    helloFromGPU<<<2, 10>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}