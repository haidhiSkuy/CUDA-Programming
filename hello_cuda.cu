#include "hip/hip_runtime.h"

#include <stdio.h> 

__global__ void helloFromGPU()
{ 
    printf("Hello CUDA\n"); 
}

int main() {
    // 2 blocks, 10 threads
    helloFromGPU<<<2, 10>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}