#include "hip/hip_runtime.h"

#include <stdio.h>  

__global__ void printThreadIds()
{ 
    printf("threadIdx.x %d, threadIdx.y: %d, threadIdx.z: %d\n", 
            threadIdx.x, threadIdx.y, threadIdx.z
    ); 
} 

int main()
{ 
    int nx, ny; 
    nx = 16; 
    ny = 16; 

    dim3 block(8, 8); 
    dim3 grid(nx / block.x, ny / block.y);  // 2, 2

    printThreadIds<<<grid, block>>>();
    
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}