#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//for random intialize
#include <stdlib.h>
#include <time.h>

//for memset
#include <cstring>

#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true){
    if (code != hipSuccess)
    {   
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void sum_arrays_gpu(int * a, int * b, int* c, int size)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < size)
		c[index] = a[index] + b[index];
}

void sum_arrays_cpu(int * a, int * b, int * c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int size = 50000000;
	int block_size = 64;

	//number of bytes needed to hold element count
	size_t NO_BYTES = size * sizeof(int);

	// host pointers
	int *h_a, *h_b, *gpu_result, *cpu_result;

	//allocate memory for host size pointers
	h_a = (int *)malloc(NO_BYTES);
	h_b = (int *)malloc(NO_BYTES);
	gpu_result = (int *)malloc(NO_BYTES);
	cpu_result = (int *)malloc(NO_BYTES);

	//initialize h_a and h_b arrays randomly
	time_t t;
	srand((unsigned)time(&t));

	for (size_t i = 0; i < size; i++)
	{
		h_a[i] = (int)(rand() & 0xFF);
		//h_a[i] = 1;
	}

	for (size_t i = 0; i < size; i++)
	{
		h_b[i] = (int)(rand() & 0xFF);
		//h_b[i] = 2;
	}

	memset(gpu_result, 0, NO_BYTES);
	memset(cpu_result, 0, NO_BYTES);

	//summation in CPU
	clock_t cpu_start, cpu_end;
	cpu_start = clock();
	sum_arrays_cpu(h_a, h_b, cpu_result, size);
	cpu_end = clock();

	int *d_a, *d_b, *d_c;
	gpuErrorCheck(hipMalloc((int **)&d_a, NO_BYTES));
	gpuErrorCheck(hipMalloc((int **)&d_b, NO_BYTES));
	gpuErrorCheck(hipMalloc((int **)&d_c, NO_BYTES));

	//kernel launch parameters
	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	clock_t mem_htod_start, mem_htod_end;
	mem_htod_start = clock();
	gpuErrorCheck(hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice));
	gpuErrorCheck(hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice));
	mem_htod_end = clock();

	//execution time measuring in GPU
	clock_t gpu_start, gpu_end;
	gpu_start = clock();

	sum_arrays_gpu << <grid, block >> > (d_a, d_b, d_c, size);
	gpuErrorCheck(hipDeviceSynchronize());
	gpu_end = clock();

	clock_t mem_dtoh_start, mem_dtoh_end;
	mem_dtoh_start = clock();
	gpuErrorCheck(hipMemcpy(gpu_result, d_c, NO_BYTES, hipMemcpyDeviceToHost));
	mem_dtoh_end = clock();

    printf("Block size : %d \n", block_size);;

	printf("CPU sum time : %4.6f \n",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

	printf("GPU kernel execution time sum time : %4.6f \n",
		(double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));

	printf("Mem transfer host to device : %4.6f \n",
		(double)((double)(mem_htod_end - mem_htod_start) / CLOCKS_PER_SEC));

	printf("Mem transfer device to host : %4.6f \n",
		(double)((double)(mem_dtoh_end - mem_dtoh_start) / CLOCKS_PER_SEC));

	printf("Total GPU time : %4.6f \n",
		(double)((double)((mem_htod_end - mem_htod_start)
			+ (gpu_end - gpu_start)
			+ (mem_dtoh_end - mem_dtoh_start)) / CLOCKS_PER_SEC));

	gpuErrorCheck(hipFree(d_a));
	gpuErrorCheck(hipFree(d_b));
	gpuErrorCheck(hipFree(d_c));

	free(h_a);
	free(h_b);
	free(gpu_result);
}
