#include "hip/hip_runtime.h"

#include <stdio.h>  

__global__ void printThreadIds()
{ 
    printf("blockIdx.x: %d | blockIdx.y: %d | blockDim.x: %d | blockDim.y: %d | gridDim.x: %d | gridDim.y: %d |\n", 
            blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y
        ); 
} 

int main()
{ 
    int nx, ny; 
    nx = 16; 
    ny = 16; 

    dim3 block(8, 8); 
    dim3 grid(nx / block.x, ny / block.y);  // 2, 2

    printThreadIds<<<grid, block>>>();
    
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}